#include "hip/hip_runtime.h"
%%writefile simple_vector.cu
#include <iostream>
#include <hip/hip_runtime.h>\
using namespace std;

#define N 1024  

__global__ void vectorAdd(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    size_t size = N * sizeof(float);

    h_A = new float[N];
    h_B = new float[N];
    h_C = new float[N];

    for (int i = 0; i < N; ++i) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    cout << "Result (first 10 values):\n";
    for (int i = 0; i < 10; ++i) {
        cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] <<endl;
    }

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}