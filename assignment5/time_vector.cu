#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;   

#define N 1024

__device__ __managed__ float A[N];
__device__ __managed__ float B[N];
__device__ __managed__ float C[N];

__global__ void vectorAddStatic() {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    for (int i = 0; i < N; ++i) {
        A[i] = static_cast<float>(i);
        B[i] = static_cast<float>(i * 2);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);
    vectorAddStatic<<<blocksPerGrid, threadsPerBlock>>>();
    hipEventRecord(stop);

    hipEventSynchronize(stop);  

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Kernel execution time: " << milliseconds << " ms\n";

    cout << "Vector addition result (first 5 values):\n";
    for (int i = 0; i < 5; ++i) {
        cout << A[i] << " + " << B[i] << " = " << C[i] << "\n";
    }

    return 0;
}